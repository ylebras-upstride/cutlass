#include <iostream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/convolution.h"
#include "cutlass/util/tensor_view_io.h"

#include "cutlass/quaternion.h"

#include "helper.h"

template<typename ElementType>
struct Conv2DOp {

  // The code section below describes datatype for input, output tensors and computation between
  // elements
  using ElementAccumulator = ElementType;        // Data type of accumulator
  using ElementComputeEpilogue = ElementType;    // Data type of epilogue computation (alpha, beta)
  using ElementInputA = ElementType;             // Data type of elements in input tensor
  using ElementInputB = ElementType;             // Data type of elements in input tensor
  using ElementOutput = ElementType;             // Data type of elements in output tensor

  using LayoutInputA = cutlass::layout::TensorNHWC;
  using LayoutInputB = cutlass::layout::TensorNHWC;
  using LayoutOutput = cutlass::layout::TensorNHWC;

  // This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
  using MMAOp = cutlass::arch::OpClassSimt;

  // This code section describes CUDA SM architecture number
  using SmArch = cutlass::arch::Sm75;

  // This code section describes the tile size a thread block will compute
  using ThreadblockShape = cutlass::gemm::GemmShape<64, 64, 8>;  // Threadblock tile shape

  // This code section describes tile size a warp will compute
  using WarpShape = cutlass::gemm::GemmShape<64, 64, 8>;         // Warp tile shape

  // This code section describes the size of MMA op
  using InstructionShape = cutlass::gemm::GemmShape<1, 1, 1>;    // TensorCore instruction shape

  // This code section describes how threadblocks are scheduled on GPU
  using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

  // Number of pipelines you want to use
  static const int NumStages = 2;

  // This code section describes the epilogue part of the kernel, we use default value
  using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
      ElementOutput,                                     // Data type of output matrix.
      1,                                                 // The number of elements per vectorized.
                                                        // memory access. This becomes the vector width of
                                                        // math instructions in the epilogue too.
      ElementAccumulator,                                // Data type of accumulator
      ElementComputeEpilogue>;                           // Data type for alpha/beta in linear combination


  using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
    ElementInputA, LayoutInputA,
    ElementInputB, LayoutInputB,
    ElementOutput, LayoutOutput,
    ElementAccumulator,
    MMAOp,
    SmArch,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    SwizzleThreadBlock,
    NumStages,
    cutlass::arch::OpMultiplyAddSaturate,
    cutlass::conv::IteratorAlgorithm::kAnalytic
  >::Kernel;

  using ImplicitGemm = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;
};


/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;
  bool use_quaternions;
  bool reference_check;
  bool measure_performance;
  int iterations;
  bool save_workspace;
  float alpha;
  float beta;
  bool benchmark;
  std::string tag;

  Options():
    help(false),
    input_size(1, 32, 32, 32),
    filter_size(32, 3, 3, 32),
    padding(1, 1, 1, 1),
    conv_stride(1, 1),
    dilation(1, 1),
    use_quaternions(false),
    reference_check(false),
    measure_performance(true),
    iterations(20),
    save_workspace(false),
    alpha(1),
    beta(0),
    benchmark(false) { }

  // Verify the problem size is compatible with the CUTLASS Convolution implementation.
  bool valid() {

    //
    // CUTLASS attempts to load 128b vectors of int4b_t elements. Consequently,
    // all pointers, strides, and tensor extents must be divisible by 32 elements.
    //
    int const kAlignment = 32;

    if ((input_size.c() % kAlignment) ||
      (filter_size.n() % kAlignment)) {

      // misaligned tensors
      return false;
    }

    // Invalid padding
    if ((padding.h() != filter_size.h() / 2) ||
      (padding.w() != filter_size.w() / 2)) {

      return false;
    }

    return true;
  }

  /// Updates input and filter sizes
  void update(
    cutlass::Tensor4DCoord input_size,
    cutlass::Tensor4DCoord filter_size) {

    this->input_size = input_size;
    this->filter_size = filter_size;

    padding.n() = filter_size.h() / 2;
    padding.h() = filter_size.h() / 2;
    padding.w() = filter_size.w() / 2;
    padding.c() = filter_size.w() / 2;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    if (cmd.check_cmd_line_flag("benchmark")) {
      benchmark = true;
    }

    if (cmd.check_cmd_line_flag("quat")) {
      use_quaternions = true;
    }

    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());

    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());
    filter_size.c() = input_size.c(); 

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);

    if (filter_size.h() == 3 && filter_size.w() == 3) {
      padding = {1, 1, 1, 1};
    }
    else {
      filter_size.h() = 1;
      filter_size.w() = 1;
      padding = {0, 0, 0, 0};
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "18_turing_simt_conv2dfprop example\n\n"
      << "  This example computes forward 2D convolution of two floating-point or quaternion NHWC tensors.\n\n"
      << "Options:\n\n"
      << "  --help               If specified, displays this usage statement.\n\n"
      << "  --n <int>            Input tensor extent N\n"
      << "  --h <int>            Input tensor extent H\n"
      << "  --w <int>            Input tensor extent W\n"
      << "  --c <int>            Input tensor extent C\n"
      << "  --k <int>            Filter extent K\n"
      << "  --r <int>            Filter extent R\n"
      << "  --s <int>            Filter extent S\n\n"
      << "  --alpha <float>      Epilogue scalar alpha\n"
      << "  --beta <float>       Epilogue scalar beta\n\n"
      << "  --quat               If set (true), tensor entries are quaternions, scalars otherwise."
      << "  --ref-check          If set (true), reference check on the host is computed\n"
      << "  --perf-check         If set (true), performance is measured.\n"
      << "  --benchmark          If set (true), performance benchmarking on several layers and batch-size.\n"
      << "  --iterations <int>   Number of profiling iterations to perform.\n"
      << "  --save-workspace     If set, workspace is written to a text file.\n"
      << "  --tag <string>       String to replicate across the first column in the results table\n";

    out << "\n\nExamples:\n\n"
      << "$ ./examples/18_turing_simt_conv2dfprop/18_turing_simt_conv2dfprop  --n=32 --h=224 --w=224 --c=128 --k=256 --r=1 --s=1\n\n"
      << "$ ./examples/18_turing_simt_conv2dfprop/18_turing_simt_conv2dfprop  --n=1 --h=224 --w=224 --c=32 --k=32 --r=3 --s=3 --ref-check\n\n";

    return out;
  }
  
  /// Computes the output tensor size (NPQK)
  cutlass::Tensor4DCoord output_size() const {
    return cutlass::Tensor4DCoord(
      input_size.n(),
      (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
      (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
      filter_size.n());
  }

  /// Defines a elementwise multiplication cost
  template<typename ElementType>
  int64_t gflops_multiplier() const;
 
  /// Compute performance in GFLOP/s
  template<typename ElementType>
  double gflops(double runtime_s) const {

    // Number of multiply-adds = NPQK * CRS
    int64_t fmas = gflops_multiplier<ElementType>() * output_size().product() * int64_t(filter_size.h() * filter_size.w() * filter_size.c());
    
    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

template<>
int64_t Options::gflops_multiplier<float>() const {
  return 1;
}

template<>
int64_t Options::gflops_multiplier<cutlass::Quaternion<float>>() const {
  return 16;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Result {
  double runtime_ms;
  double gflops;
  cutlass::Status status;
  cutlass::Status reference_check;
  hipError_t error;

  Result(): 
    runtime_ms(0), 
    gflops(0),
    status(cutlass::Status::kSuccess),
    reference_check(cutlass::Status::kInvalid),
    error(hipSuccess) { }

  static std::ostream & print_header(std::ostream &out, Options const &options) {

    if (!options.tag.empty()) {
      out << "Name,";
    }

    out << "Layer,N,H,W,C,K,R,S,Runtime,GFLOPs";

    return out;
  }

  std::ostream & print(std::ostream &out, int idx, Options const &options) {

    if (!options.tag.empty()) {
      out << options.tag << ",";
    }

    out 
      << "conv_" << idx << ","
      << options.input_size.n() << ","
      << options.input_size.h() << ","
      << options.input_size.w() << ","
      << options.input_size.c() << ","
      << options.filter_size.n() << ","
      << options.filter_size.h() << ","
      << options.filter_size.w() << ","
      << runtime_ms << ","
      << gflops;

    return out;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

template<typename ElementType>
ElementType make_element_from_float(float value);

template<>
float make_element_from_float(float value) {
  return value;
}

template<>
cutlass::Quaternion<float> make_element_from_float(float value) {
  return cutlass::Quaternion<float>(value, value, value, value);
}

/// Runs one benchmark
template<typename ElementType>
Result profile_convolution(Options const &options) {

  Result result;

  //
  // Allocate host-device tensors using the CUTLASS Utilities.
  //

  using ElementInputA = typename Conv2DOp<ElementType>::ElementInputA;
  using ElementInputB = typename Conv2DOp<ElementType>::ElementInputB;
  using ElementOutput = typename Conv2DOp<ElementType>::ElementOutput;
  using ElementOutput = typename Conv2DOp<ElementType>::ElementOutput;
  using LayoutInputA = typename Conv2DOp<ElementType>::LayoutInputA;
  using LayoutInputB = typename Conv2DOp<ElementType>::LayoutInputB;
  using LayoutOutput = typename Conv2DOp<ElementType>::LayoutOutput;
  using LayoutOutput = typename Conv2DOp<ElementType>::LayoutOutput;
  using ElementComputeEpilogue = typename Conv2DOp<ElementType>::ElementComputeEpilogue;
  using ElementAccumulator = typename Conv2DOp<ElementType>::ElementAccumulator;

  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(options.input_size);
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(options.filter_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(options.output_size());
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_c(options.output_size());

  //
  // Initialize tensors
  //

  // Fill tensor A on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      make_element_from_float<ElementInputA>(7),
      make_element_from_float<ElementInputA>(-8),
      0);

  // Fill tensor B on host with uniform-distribution random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      1,
      make_element_from_float<ElementInputB>(7),
      make_element_from_float<ElementInputB>(-8),
      0);

  // Fill tensor C on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_c.host_view());

  // Fill tensor C for reference on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_ref_c.host_view());

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c.sync_device();
  tensor_ref_c.sync_device();

  //
  // Define arguments for CUTLASS Convolution
  //

  // mode (kCrossCorrelation or kConvolution)
  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Construct Conv2dProblemSize with user defined output size
  cutlass::conv::Conv2dProblemSize problem_size(      
      options.input_size,
      options.filter_size,
      options.padding,
      options.conv_stride,
      options.dilation,
      options.output_size(),
      mode,
      split_k_slices);

  // Construct ImplicitGemm::Argument structure with conv2d 
  // problem size, data pointers, and epilogue values
  using ImplicitGemm = typename Conv2DOp<ElementType>::ImplicitGemm;
  typename ImplicitGemm::Arguments arguments{
    problem_size,
    tensor_a.device_ref(),
    tensor_b.device_ref(),
    tensor_c.device_ref(),
    tensor_c.device_ref(),
    {options.alpha, options.beta},
  };

  //
  // Initialize CUTLASS Convolution
  //

  ImplicitGemm implicit_gemm_op;

  size_t workspace_size = implicit_gemm_op.get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  result.status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(result.status);

  result.status = implicit_gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(result.status);

  //
  // Launch initialized CUTLASS kernel
  //
  result.status = implicit_gemm_op();

  CUTLASS_CHECK(result.status);

  //
  // Optional reference check
  //
  
  if (options.reference_check) {
    std::cout << "Verification on host...\n";

    // Compute with reference implementation
    cutlass::reference::host::Conv2dFprop<
      ElementInputA,
      LayoutInputA,
      ElementInputB,
      LayoutInputB,
      ElementOutput,
      LayoutOutput,
      ElementComputeEpilogue,
      ElementAccumulator,
      cutlass::NumericConverter<ElementOutput, ElementComputeEpilogue>
    >(
      problem_size,
      tensor_a.host_ref(),
      tensor_b.host_ref(),
      tensor_c.host_ref(),
      tensor_ref_c.host_ref(),
      options.alpha,
      options.beta
    );

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    tensor_c.sync_host();

    bool passed = cutlass::reference::host::TensorEquals(
      tensor_c.host_view(),
      tensor_ref_c.host_view());

    if (!passed) {
      result.reference_check = cutlass::Status::kErrorInternal;
      std::cout << "ERROR - results miscompared.\n";
    }
    else {
      result.reference_check = cutlass::Status::kSuccess;
      std::cout << "Passed.\n";
    }
  }
  else {
    result.reference_check = cutlass::Status::kInvalid;
  }

  if (options.save_workspace) {

    std::stringstream ss;

    ss << "19_tensor_conv_workspace_conv2dfprop_f32_"
      << options.input_size.n() << "x" << options.input_size.h() << "x" << options.input_size.w() << "x" << options.input_size.c() 
      << "_"
      << options.filter_size.n() << "x" << options.filter_size.h() << "x" << options.filter_size.w() << "x" << options.filter_size.c() 
      << ".dat";

    std::ofstream output_workspace(ss.str());

    output_workspace 
      << "Input = \n" << tensor_a.host_view() << "\n\n"
      << "Filters = \n" << tensor_b.host_view() << "\n\n";

    if (options.reference_check) {
      output_workspace << "Reference = \n" << tensor_ref_c.host_view() << "\n\n";
    }

    output_workspace << "Computed = \n" << tensor_c.host_view() << std::endl;

    std::cout << "Results written to '" << ss.str() << "'." << std::endl;
  }
  
  //
  // Performance measurement
  //

  if (options.measure_performance) {

    // update tensor contents on GPU to avoid impact on runtime (happens with --ref-check option)
    tensor_a.sync_device();
    tensor_b.sync_device();
    tensor_c.sync_device();

    hipEvent_t events[2];
    
    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return result;
      }
    }

    // Record an event at the start of a series of convolution operations.
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Launch a sequence of implicit GEMM operations on the device
    for (int iteration = 0; iteration < options.iterations; ++iteration) {
      result.status = implicit_gemm_op();
      CUTLASS_CHECK(result.status);
    }

    // Record an event when the convolutions have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Print average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops<ElementType>(result.runtime_ms / 1000.0);

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.benchmark) {
    // Benchmark several layers

    int batch_sizes[] = {1, 32, 64, 128, 256, 512};

    struct Benchmark {
      int h, w, c, k, r, s;
    } layers[] = {
      {56,  56,   64,   256, 1, 1},
      {56,  56,   64,    64, 1, 1},
      {56,  56,   64,    64, 3, 3},
      {56,  56,  256,    64, 1, 1},
      {56,  56,  256,   512, 1, 1},
      {56,  56,  256,   128, 1, 1},
      {28,  28,  128,   128, 3, 3},
      {28,  28,  128,   512, 1, 1},
      {28,  28,  512,   128, 1, 1},
      {28,  28,  512,  1024, 1, 1},
      {28,  28,  512,   256, 1, 1},
      {14,  14,  256,   256, 3, 3},
      {14,  14,  256,  1024, 1, 1},
      {14,  14,  1024,  256, 1, 1},
      {14,  14,  1024, 2048, 1, 1},
      {14,  14,  1024,  512, 1, 1},
      {7,    7,   512,  512, 3, 3},
    };

    Result::print_header(std::cout, options) << std::endl;

    int idx = 1;

    for (auto const &layer : layers) {
      for (auto N : batch_sizes) {

        options.update({N, layer.h, layer.w, layer.c}, {layer.k, layer.r, layer.s, layer.c});

        Result result;
        if (options.use_quaternions) {
          result = profile_convolution<cutlass::Quaternion<float>>(options);
        }
        else {
          result = profile_convolution<float>(options);
        }
        result.print(std::cout, idx, options) << std::endl;
      }

      ++idx;
    }
  }
  else {

    // Execute one problem size
    if (!options.valid()) {
      std::cerr << "Invalid problem." << std::endl;
      return -1;
    }
    
    Result result;
    if (options.use_quaternions) {
      result = profile_convolution<cutlass::Quaternion<float>>(options);
    }
    else {
      result = profile_convolution<float>(options);
    }

    Result::print_header(std::cout, options) << std::endl;
    result.print(std::cout, 1, options) << std::endl;
  }

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////